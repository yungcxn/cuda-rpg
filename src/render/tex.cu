#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "tex.h"
#include "../res/res.h"

static tex_tileline_t* devtilemap;
static tex_realrgba_t* devpalette;

static void _devtilemap_alloc() {
        const tex_tileline_t hosttilemap[] = { RES_MOTHERSHEET_DATA };
        hipMalloc(&devtilemap, sizeof(hosttilemap));
        hipMemcpy(devtilemap, hosttilemap, sizeof(hosttilemap), hipMemcpyHostToDevice);
}

static void _devpalette_alloc() {
        const tex_realrgba_t hostpalette[] = { RES_PALETTE_DATA };
        hipMalloc(&devpalette, sizeof(hostpalette));
        hipMemcpy(devpalette, hostpalette, sizeof(hostpalette), hipMemcpyHostToDevice);
}

void tex_devdata_init() {
        _devtilemap_alloc();
        _devpalette_alloc();
}

static void _devtilemap_free() {
        hipFree(devtilemap);
        devtilemap = 0;
}

static void _devpalette_free() {
        hipFree(devpalette);
        devpalette = 0;
}

void tex_devdata_cleanup() {
        _devtilemap_free();
        _devpalette_free();
}