#include "cudamem.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>

extern "C" {

uint32_t cudamem_alloc(void** ptr, size_t bytes) {
        hipError_t err = hipMalloc(ptr, bytes);
        if (err != hipSuccess) {
                std::fprintf(stderr, "cudamem_alloc: %s\n", hipGetErrorString(err));
                *ptr = nullptr;
                return -1;
        }
        return 0;
}

uint32_t cudamem_free(void* ptr) {
        if (!ptr) return 0;
        hipError_t err = hipFree(ptr);
        if (err != hipSuccess) {
                std::fprintf(stderr, "cudamem_free: %s\n", hipGetErrorString(err));
                return -1;
        }
        return 0;
}

uint32_t cudamem_copy(void* dst, const void* src, size_t bytes, uint32_t direction) {
        hipMemcpyKind kind;
        switch (direction) {
                case 0: kind = hipMemcpyHostToHost; break;
                case 1: kind = hipMemcpyHostToDevice; break;
                case 2: kind = hipMemcpyDeviceToHost; break;
                case 3: kind = hipMemcpyDeviceToDevice; break;
                default: return -1;
        }
        hipError_t err = hipMemcpy(dst, src, bytes, kind);
        if (err != hipSuccess) {
                std::fprintf(stderr, "cudamem_copy: %s\n", hipGetErrorString(err));
                return -1;
        }
        return 0;
}

uint32_t cudamem_move(void** dst, const void* src, size_t bytes, uint32_t direction) {
        void* new_ptr = nullptr;
        if (cudamem_alloc(&new_ptr, bytes) != 0)
                return -1;
        if (cudamem_copy(new_ptr, src, bytes, direction) != 0) {
                cudamem_free(new_ptr);
                return -1;
        }
        if (*dst)
                cudamem_free(*dst);
        *dst = new_ptr;
        return 0;
}

}
