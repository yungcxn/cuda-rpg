#include "hip/hip_runtime.h"

#include "kernel.cuh"
#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ static void _fill_chessboard_kernel(hipSurfaceObject_t surface, int width, int height) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < width && y < height) {
                bool is_white = ((x + y) % 2 == 0);

                uchar4 color;
                if (is_white) {
                        color = make_uchar4(255, 255, 255, 255);
                } else {
                        color = make_uchar4(0, 0, 0, 255);
                }

                surf2Dwrite(color, surface, x * sizeof(uchar4), y);
        }
}

void kernel_draw(hipSurfaceObject_t surf, uint32_t w, uint32_t h) {
        /* per-pixel kernel */
        dim3 block(16, 16);
        dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);

        _fill_chessboard_kernel<<<grid, block>>>(surf, w, h);
}